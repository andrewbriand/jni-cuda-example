#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <jni.h>

#include "VectorAdd.h"

#define CHK_CUDA(expression)                                                  \
  {                                                                           \
    hipError_t status = (expression);                                        \
    if (status != hipSuccess) {                                              \
      std::cerr << "Error in file: " << __FILE__ << ", on line: " << __LINE__ \
                << ": " << hipGetErrorString(status) << std::endl;           \
      std::exit(EXIT_FAILURE);                                                \
    }                                                                         \
  }

__global__ void vector_add_kernel(int *a, int *b, int *c, size_t len) {

  size_t offset = blockIdx.x * blockDim.x + threadIdx.x;

  if (offset < len) {
    c[offset] = a[offset] + b[offset];
  }

}

JNIEXPORT jintArray JNICALL Java_VectorAdd_add(JNIEnv *env, jclass thisClass,
		                               jintArray a, jintArray b) {

  size_t len = env->GetArrayLength(a);

  int *h_a = env->GetIntArrayElements(a, nullptr);
  int *h_b = env->GetIntArrayElements(b, nullptr);
  std::vector<int> h_c(len);

  int *d_a, *d_b, *d_c;

  CHK_CUDA(hipMalloc(&d_a, sizeof(int) * len));
  CHK_CUDA(hipMalloc(&d_b, sizeof(int) * len));
  CHK_CUDA(hipMalloc(&d_c, sizeof(int) * len));

  CHK_CUDA(hipMemcpy(d_a, h_a, sizeof(int) * len, hipMemcpyHostToDevice));
  CHK_CUDA(hipMemcpy(d_b, h_b, sizeof(int) * len, hipMemcpyHostToDevice));

  const int block_size = 32;
  const int num_blocks = (len + block_size - 1) / block_size;

  vector_add_kernel<<<num_blocks, block_size>>>(d_a, d_b, d_c, len);

  CHK_CUDA(hipGetLastError());

  CHK_CUDA(hipMemcpy(h_c.data(), d_c, sizeof(int) * len, hipMemcpyDeviceToHost));

  CHK_CUDA(hipFree(d_a));
  CHK_CUDA(hipFree(d_b));
  CHK_CUDA(hipFree(d_c));

  env->ReleaseIntArrayElements(a, h_a, 0);
  env->ReleaseIntArrayElements(b, h_b, 0);

  jintArray c = env->NewIntArray(len);

  env->SetIntArrayRegion(c, 0, len, h_c.data());

  return c;
}
