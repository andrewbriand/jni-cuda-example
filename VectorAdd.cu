#include "hip/hip_runtime.h"
#include <vector>

#include <jni.h>

#include "VectorAdd.h"

__global__ void vector_add_kernel(int *a, int *b, int *c, size_t len) {

  size_t offset = blockIdx.x * blockDim.x + threadIdx.x;

  if (offset < len) {
    c[offset] = a[offset] + b[offset];
  }

}

JNIEXPORT jintArray JNICALL Java_VectorAdd_add(JNIEnv *env, jclass thisClass,
		                               jintArray a, jintArray b) {

  size_t len = env->GetArrayLength(a);

  int *h_a = env->GetIntArrayElements(a, nullptr);
  int *h_b = env->GetIntArrayElements(b, nullptr);
  std::vector<int> h_c(len);

  int *d_a, *d_b, *d_c;

  hipMalloc(&d_a, sizeof(int) * len);
  hipMalloc(&d_b, sizeof(int) * len);
  hipMalloc(&d_c, sizeof(int) * len);

  hipMemcpy(d_a, h_a, sizeof(int) * len, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, sizeof(int) * len, hipMemcpyHostToDevice);

  const int block_size = 32;
  const int num_blocks = len / block_size + 1;

  vector_add_kernel<<<num_blocks, block_size>>>(d_a, d_b, d_c, len);

  hipMemcpy(h_c.data(), d_c, sizeof(int) * len, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  env->ReleaseIntArrayElements(a, h_a, 0);
  env->ReleaseIntArrayElements(b, h_b, 0);

  jintArray c = env->NewIntArray(len);

  env->SetIntArrayRegion(c, 0, len, h_c.data());

  return c;
}
